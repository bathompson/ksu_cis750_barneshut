#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
extern "C"
{
#include "../common/vector_utils.h"
#include "../common/fileIO_util.h"
#include "../common/octree.h"
}
#include "vector_utils.cuh"
#include "cuda_octree.cuh"
#include <chrono>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

int G = 0;
int capcity = 0;

static const int blockSize = 32;

void __cudaCheck(hipError_t err, const char* file, const int line);
#define cudaCheck(err) __cudaCheck (err, __FILE__, __LINE__)

void __cudaCheckLastError(const char* errorMessage, const char* file, const int line);
#define cudaCheckLastError(msg) __cudaCheckLastError (msg, __FILE__, __LINE__)

void __cudaCheck(hipError_t err, const char *file, const int line)
{
    if( hipSuccess != err) {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

void __cudaCheckLastError(const char *errorMessage, const char *file, const int line)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
                file, line, errorMessage, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

__device__ Vec3f computeBarnesHutForce(Octree root, int node, Body body, float theta, int G) {
    //Check if the root is null, if so return 0 valued vector.
    if(node == -1) {
        return newVec3fGPU(0,0,0);
    }
    //Calculate vector values.
    double scalarForce = 0;
    Vec3f vectorDist = ptToVectorGPU(body.pos, root.massPosition[node]);
    float distSq = vectorDotGPU(vectorDist, vectorDist);
    float invDist = __frsqrt_rn(distSq);
    //Check if the body and root are equal, if so there is no forces
    if(vectorEqGPU(body.pos, root.massPosition[node])) {
        return newVec3fGPU(0, 0, 0);
        //Check either barnes hut condition or single body.
    } else if(2*root.dist[node]*invDist < theta || root.singleBody[node]) {
        //Find the force between two bodies
        scalarForce = G * body.mass * root.mass[node] / distSq;
        return vectorScalarMultGPU(scalarForce, vectorNormalizeGPU(vectorDist));
    } else {
        //Loop through all children
        Vec3f netForce = newVec3fGPU(0,0,0);
        for(size_t i = 0; i < 8; i++) {
            netForce = vectorAddGPU(computeBarnesHutForce(root, root.children[node * 8 + i], body, theta, G), netForce);
        }
        return netForce;
    }
}

__device__ void findMaxSize(Body* bodies, int bodyCount, float *blockMax) {
    cg::grid_group grid = cg::this_grid();

    // Parallel reduction inspired by https://riptutorial.com/cuda/example/22458/multi-block-parallel-reduction-for-commutative-operator
    int threadIndex = threadIdx.x;
    int gridThreadIndex = threadIdx.x + blockIdx.x * blockDim.x;
    const int gridSize = blockDim.x * gridDim.x;

    // Compute max on this block
    float max = 0.0f;
    for (int i = gridThreadIndex; i < bodyCount; i += gridSize)
    {
        max = fmaxf(max, fabsf(bodies->pos.x));
        max = fmaxf(max, fabsf(bodies->pos.y));
        max = fmaxf(max, fabsf(bodies->pos.z));
    }
    __shared__ float shArr[blockSize];
    shArr[threadIndex] = max;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2)
    {
        if (threadIndex < size)
            shArr[threadIndex] = fmaxf(shArr[threadIndex], shArr[threadIndex + size]);
        __syncthreads();
    }
    if (threadIndex == 0)
        blockMax[blockIdx.x] = shArr[0];

    grid.sync();

    // If we are the first block, compute max from other blocks
    if (gridThreadIndex == 0) {
        shArr[threadIndex] = threadIndex < gridSize ? blockMax[threadIndex] : 0;
        __syncthreads();
        for (int size = blockSize / 2; size > 0; size /= 2) {
            if (threadIndex < size)
                shArr[threadIndex] = fmaxf(shArr[threadIndex], shArr[threadIndex + size]);
            __syncthreads();
        }
        if (threadIndex == 0)
            blockMax[0] = shArr[0];
    }
}

__global__ void constructBarnesHutTree(Octree *globalTree, Body *frame, int count, int capacity, float *blockMax) {
    cg::grid_group grid = cg::this_grid();

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    Octree tree = *globalTree;

    findMaxSize(frame, count, blockMax);
    resetOctreeGPU(tree, capacity);
    grid.sync();
    tree = setDiameterGPU(tree, blockMax[0]);

    if (index == 0) {
        for (size_t i = 0; i < count; i++) {
            insertElementGPU(tree, 0, frame[i].pos, frame[i].mass);
        }
    }

    if (index == 0) {
        *globalTree = tree;
    }
}

__global__ void simulateFrame(Octree *globalTree, Body *frame, int count, float t, float theta, int G) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < count) {
        Octree tree = *globalTree;
        Body body = frame[index];
        Vec3f netForce = computeBarnesHutForce(tree, 0, body, theta, G);
        Vec3f accel = vectorScalarMultGPU(1 / body.mass, netForce);
        body.mass = body.mass;
        body.pos = finalPosGPU(accel, body.vel, body.pos, t);
        body.vel = finalVelGPU(accel, body.vel, t);
        frame[index] = body;
    }
}

int main(int argc, char **argv) {
    //Declarations
    size_t bodyCount;
    size_t timeSteps;
    FILE *data;
    FILE *out;
    Body **frames;
    Body *gpuFrame;
    Vec3f **pos;
    Octree *root;
    char *outputFileName;
    char *filePath;
    float *masses;
    float deltaT;
    float theta = 0.0f;

    //Handle input
    if(argc < 8) {
        printf("Usage: <path to input> <number of inputs> <number of timesteps> <length of time steps in seconds> <theta> <G> <outputFileName>\n");
        exit(0);
    }

    filePath = argv[1];
    bodyCount = strtoul(argv[2], NULL, 10);
    timeSteps = strtoul(argv[3], NULL, 10);
    deltaT = atof(argv[4]);
    theta = atof(argv[5]);
    G = atoi(argv[6]);
    outputFileName = argv[7];

    //Allocate Memory
    frames = (Body **) malloc(timeSteps * sizeof(Body *));

    for(size_t i = 0; i < timeSteps; i++) {
        frames[i] = (Body *) malloc(bodyCount * sizeof(Body));
    }

    hipMalloc(&gpuFrame, bodyCount * sizeof(Body));

    masses = (float *) malloc(bodyCount * sizeof(float));
    pos = (Vec3f **) malloc(timeSteps * sizeof(Vec3f *));
    for(size_t i = 0; i < timeSteps; i++) {
        pos[i] = (Vec3f *) malloc(bodyCount * sizeof(Vec3f));
    }

    capcity = bodyCount * 2;

    Octree allocatedTree = allocateOctreeCUDA(capcity);
    Octree *gpuTree = nullptr;
    hipMalloc(&gpuTree, sizeof(Octree));
    hipMemcpy(gpuTree, &allocatedTree, sizeof(Octree), hipMemcpyHostToDevice);

    float *gpuBlockMax = nullptr;
    hipMalloc(&gpuBlockMax, sizeof(float));

    //Data input
    data = fopen(filePath, "r");
    readInput(data, frames[0], bodyCount);
    fclose(data);

    // Begin timing
    auto starttime = std::chrono::high_resolution_clock::now();

    // Copy initial frame of bodies to CUDA
    hipMemcpy(gpuFrame, frames[0], bodyCount * sizeof(Body), hipMemcpyHostToDevice);

    //Do the thing
    for(size_t i = 0; i < timeSteps - 1; i++) {
        // Build tree on GPU
        void *kernelArgs[] = {&gpuTree, &gpuFrame, &bodyCount, &capcity, &gpuBlockMax };
        hipLaunchCooperativeKernel((void*)&constructBarnesHutTree, bodyCount, blockSize, kernelArgs);
        hipDeviceSynchronize();

        // Simulate frame on GPU
        simulateFrame<<<(bodyCount + (blockSize - 1)) / blockSize, blockSize>>>(gpuTree, gpuFrame, bodyCount, deltaT, theta, G);
        hipDeviceSynchronize();

        // Copy results back
        hipMemcpy(frames[i + 1], gpuFrame, bodyCount * sizeof(Body), hipMemcpyDeviceToHost);
    }

    // End timing
    auto endtime = std::chrono::high_resolution_clock::now();
    double runtime = std::chrono::duration_cast<std::chrono::milliseconds>(endtime - starttime).count() / 1000.0;
    printf("Simulated %d frames in %.4f seconds\n", timeSteps, runtime);

    //Put data into output format
    for(size_t i = 0; i < bodyCount; i++) {
        masses[i] = frames[0][i].mass;
    }

    for(size_t i = 0; i < timeSteps; i++) {
        for(size_t j = 0; j < bodyCount; j++) {
            pos[i][j] = frames[i][j].pos;
        }
    }

    //Write to output
    out = fopen(outputFileName, "wb");
    writeOutput(out, bodyCount, timeSteps, masses, pos);
    fclose(out);

    //Free memory
    for(size_t i = 0; i < timeSteps; i++) {
        free(frames[i]);
        free(pos[i]);
    }

    free(frames);
    free(pos);
    free(masses);

    hipFree(gpuFrame);
    freeTreeCUDA(allocatedTree);
    hipFree(gpuTree);
    hipFree(gpuBlockMax);
}
