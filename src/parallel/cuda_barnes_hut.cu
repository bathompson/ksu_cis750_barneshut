#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
extern "C"
{
#include "../common/vector_utils.h"
#include "../common/fileIO_util.h"
#include "../common/octree.h"
}
#include "vector_utils.cuh"
#include "cuda_octree.cuh"
#include <chrono>

int G = 0;
int capcity = 0;

void __cudaCheck(hipError_t err, const char* file, const int line);
#define cudaCheck(err) __cudaCheck (err, __FILE__, __LINE__)

void __cudaCheckLastError(const char* errorMessage, const char* file, const int line);
#define cudaCheckLastError(msg) __cudaCheckLastError (msg, __FILE__, __LINE__)

void __cudaCheck(hipError_t err, const char *file, const int line)
{
    if( hipSuccess != err) {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

void __cudaCheckLastError(const char *errorMessage, const char *file, const int line)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
                file, line, errorMessage, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

__device__ Vec3f computeBarnesHutForce(Octree root, int node, Body body, float theta, int G) {
    //Check if the root is null, if so return 0 valued vector.
    if(node == -1) {
        return newVec3fGPU(0,0,0);
    }
    //Calculate vector values.
    double scalarForce = 0;
    Vec3f vectorDist = ptToVectorGPU(body.pos, root.massPosition[node]);
    float distSq = vectorDotGPU(vectorDist, vectorDist);
    float invDist = __frsqrt_rn(distSq);
    //Check if the body and root are equal, if so there is no forces
    if(vectorEqGPU(body.pos, root.massPosition[node])) {
        return newVec3fGPU(0, 0, 0);
        //Check either barnes hut condition or single body.
    } else if(2*root.dist[node]*invDist < theta || root.singleBody[node]) {
        //Find the force between two bodies
        scalarForce = G * body.mass * root.mass[node] / distSq;
        return vectorScalarMultGPU(scalarForce, vectorNormalizeGPU(vectorDist));
    } else {
        //Loop through all children
        Vec3f netForce = newVec3fGPU(0,0,0);
        for(size_t i = 0; i < 8; i++) {
            netForce = vectorAddGPU(computeBarnesHutForce(root, root.children[node * 8 + i], body, theta, G), netForce);
        }
        return netForce;
    }
}

__device__ float findMaxSize(Body* bodies, int bodyCount) {
    float max = 0.0f;
    for(size_t i = 0; i < bodyCount; i++) {
        if(fabsf(bodies->pos.x) > max) {
            max = fabsf(bodies->pos.x);
        }
        if(fabsf(bodies->pos.y) > max) {
            max = fabsf(bodies->pos.y);
        }
        if(fabsf(bodies->pos.z) > max) {
            max = fabsf(bodies->pos.z);
        }
    }
    return max;
}

__global__ void constructBarnesHutTree(Octree *globalTree, Body *frame, int count, int capacity) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index == 0) {
        Octree tree = *globalTree;
        resetOctreeGPU(tree, capacity);
        setDiameterGPU(tree, findMaxSize(frame, count));
        for (size_t i = 0; i < count; i++) {
            insertElementGPU(tree, 0, frame[i].pos, frame[i].mass);
        }
        *globalTree = tree;
    }
}

__global__ void simulateFrame(Octree *globalTree, Body *frame, int count, float t, float theta, int G) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < count) {
        Octree tree = *globalTree;
        Body body = frame[index];
        Vec3f netForce = computeBarnesHutForce(tree, 0, body, theta, G);
        Vec3f accel = vectorScalarMultGPU(1 / body.mass, netForce);
        body.mass = body.mass;
        body.pos = finalPosGPU(accel, body.vel, body.pos, t);
        body.vel = finalVelGPU(accel, body.vel, t);
        frame[index] = body;
    }
}

int main(int argc, char **argv) {
    //Declarations
    size_t bodyCount;
    size_t timeSteps;
    FILE *data;
    FILE *out;
    Body **frames;
    Body *gpuFrame;
    Vec3f **pos;
    Octree *root;
    char *outputFileName;
    char *filePath;
    float *masses;
    float deltaT;
    float theta = 0.0f;

    //Handle input
    if(argc < 8) {
        printf("Usage: <path to input> <number of inputs> <number of timesteps> <length of time steps in seconds> <theta> <G> <outputFileName>\n");
        exit(0);
    }

    filePath = argv[1];
    bodyCount = strtoul(argv[2], NULL, 10);
    timeSteps = strtoul(argv[3], NULL, 10);
    deltaT = atof(argv[4]);
    theta = atof(argv[5]);
    G = atoi(argv[6]);
    outputFileName = argv[7];

    //Allocate Memory
    frames = (Body **) malloc(timeSteps * sizeof(Body *));

    for(size_t i = 0; i < timeSteps; i++) {
        frames[i] = (Body *) malloc(bodyCount * sizeof(Body));
    }

    hipMalloc(&gpuFrame, bodyCount * sizeof(Body));

    masses = (float *) malloc(bodyCount * sizeof(float));
    pos = (Vec3f **) malloc(timeSteps * sizeof(Vec3f *));
    for(size_t i = 0; i < timeSteps; i++) {
        pos[i] = (Vec3f *) malloc(bodyCount * sizeof(Vec3f));
    }

    capcity = bodyCount * 2;

    Octree allocatedTree = allocateOctreeCUDA(capcity);
    Octree *gpuTree = nullptr;
    hipMalloc(&gpuTree, sizeof(Octree));
    hipMemcpy(gpuTree, &allocatedTree, sizeof(Octree), hipMemcpyHostToDevice);

    //Data input
    data = fopen(filePath, "r");
    readInput(data, frames[0], bodyCount);
    fclose(data);

    // Begin timing
    auto starttime = std::chrono::high_resolution_clock::now();

    // Copy initial frame of bodies to CUDA
    hipMemcpy(gpuFrame, frames[0], bodyCount * sizeof(Body), hipMemcpyHostToDevice);

    //Do the thing
    for(size_t i = 0; i < timeSteps - 1; i++) {
        // Build tree on GPU
        constructBarnesHutTree<<<1, 1>>>(gpuTree, gpuFrame, bodyCount, capcity);
        hipDeviceSynchronize();

        // Simulate frame on GPU
        simulateFrame<<<(bodyCount + 255) / 256, 256>>>(gpuTree, gpuFrame, bodyCount, deltaT, theta, G);
        hipDeviceSynchronize();

        // Copy results back
        //hipMemcpy(frames[i + 1], gpuFrame, bodyCount * sizeof(Body), hipMemcpyDeviceToHost);
    }

    // End timing
    auto endtime = std::chrono::high_resolution_clock::now();
    double runtime = std::chrono::duration_cast<std::chrono::milliseconds>(endtime - starttime).count() / 1000.0;
    printf("Simulated %d frames in %.4f seconds\n", timeSteps, runtime);

    //Put data into output format
    for(size_t i = 0; i < bodyCount; i++) {
        masses[i] = frames[0][i].mass;
    }

    for(size_t i = 0; i < timeSteps; i++) {
        for(size_t j = 0; j < bodyCount; j++) {
            pos[i][j] = frames[i][j].pos;
        }
    }

    //Write to output
    out = fopen(outputFileName, "wb");
    writeOutput(out, bodyCount, timeSteps, masses, pos);
    fclose(out);

    //Free memory
    for(size_t i = 0; i < timeSteps; i++) {
        free(frames[i]);
        free(pos[i]);
    }

    free(frames);
    free(pos);
    free(masses);

    hipFree(gpuFrame);
    freeTreeCUDA(allocatedTree);
    hipFree(gpuTree);
}