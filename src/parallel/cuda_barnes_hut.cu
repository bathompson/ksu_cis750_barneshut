#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
extern "C"
{
#include "../common/vector_utils.h"
#include "../common/fileIO_util.h"
#include "../common/octree.h"
#include "cuda_octree.h"
}
#include "vector_utils.cuh"
#include <chrono>

int G = 0;
int capcity = 0;

__device__ Vec3f computeBarnesHutForce(Octree root, int node, Body body, float theta, int G) {
    //Check if the root is null, if so return 0 valued vector.
    if(node == -1) {
        return newVec3fGPU(0,0,0);
    }
    //Calculate vector values.
    double scalarForce = 0;
    Vec3f vectorDist = ptToVectorGPU(body.pos, root.massPosition[node]);
    float distSq = vectorDotGPU(vectorDist, vectorDist);
    float invDist = 1.0f/sqrtf(distSq);
    //Check if the body and root are equal, if so there is no forces
    if(vectorEqGPU(body.pos, root.massPosition[node])) {
        return newVec3fGPU(0, 0, 0);
        //Check either barnes hut condition or single body.
    } else if(2*root.dist[node]*invDist < theta || root.singleBody[node]) {
        //Find the force between two bodies
        scalarForce = G * body.mass * root.mass[node] / distSq;
        return vectorScalarMultGPU(scalarForce, vectorNormalizeGPU(vectorDist));
    } else {
        //Loop through all children
        Vec3f netForce = newVec3fGPU(0,0,0);
        for(size_t i = 0; i < 8; i++) {
            netForce = vectorAddGPU(computeBarnesHutForce(root, root.children[node * 8 + i], body, theta, G), netForce);
        }
        return netForce;
    }
}

__global__ void simulateFrame(Octree tree, Body* frame, int count, float t, float theta, int G) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < count) {
        Body body = frame[index];
        Vec3f netForce = computeBarnesHutForce(tree, 0, body, theta, G);
        Vec3f accel = vectorScalarMultGPU(1 / body.mass, netForce);
        body.mass = body.mass;
        body.pos = finalPosGPU(accel, body.vel, body.pos, t);
        body.vel = finalVelGPU(accel, body.vel, t);
        frame[index] = body;
    }
}

float findMaxSize(Body* bodies, int bodyCount) {
    float max = 0.0f;
    for(size_t i = 0; i < bodyCount; i++) {
        if(fabs(bodies->pos.x > max)) {
            max = fabs(bodies->pos.x);
        }
        if(fabs(bodies->pos.y > max)) {
            max = fabs(bodies->pos.y);
        }
        if(fabs(bodies->pos.z > max)) {
            max = fabs(bodies->pos.z);
        }
    }
    return max;
}

void constructBarnesHutTree(Octree tree, Body *frame, size_t count) {
    resetOctreeCPU(tree, capcity);
    setDiameter(findMaxSize(frame, count));
    for(size_t i = 0; i<count; i++) {
        insertElement(tree, 0, frame[i].pos, frame[i].mass);
    }
    //_debugPrint(tree, 0, 2);
}

int main(int argc, char **argv) {
    //Declarations
    size_t bodyCount;
    size_t timeSteps;
    FILE *data;
    FILE *out;
    Body **frames;
    Body *gpuFrame;
    Vec3f **pos;
    Octree *root;
    char *outputFileName;
    char *filePath;
    float *masses;
    float deltaT;
    float theta = 0.0f;

    //Handle input
    if(argc < 8) {
        printf("Usage: <path to input> <number of inputs> <number of timesteps> <length of time steps in seconds> <theta> <G> <outputFileName>\n");
        exit(0);
    }

    filePath = argv[1];
    bodyCount = strtoul(argv[2], NULL, 10);
    timeSteps = strtoul(argv[3], NULL, 10);
    deltaT = atof(argv[4]);
    theta = atof(argv[5]);
    G = atoi(argv[6]);
    outputFileName = argv[7];

    //Allocate Memory
    frames = (Body **) malloc(timeSteps * sizeof(Body *));

    for(size_t i = 0; i < timeSteps; i++) {
        frames[i] = (Body *) malloc(bodyCount * sizeof(Body));
    }

    hipMalloc(&gpuFrame, bodyCount * sizeof(Body));

    masses = (float *) malloc(bodyCount * sizeof(float));
    pos = (Vec3f **) malloc(timeSteps * sizeof(Vec3f *));
    for(size_t i = 0; i < timeSteps; i++) {
        pos[i] = (Vec3f *) malloc(bodyCount * sizeof(Vec3f));
    }

    capcity = bodyCount * 2;
    Octree tree = allocateOctreeCPU(capcity);
    Octree gpuTree = allocateOctreeCUDA(capcity);

    //Data input
    data = fopen(filePath, "r");
    readInput(data, frames[0], bodyCount);
    fclose(data);

    // Begin timing
    auto starttime = std::chrono::high_resolution_clock::now();

    //Do the thing
    for(size_t i = 0; i < timeSteps - 1; i++) {
        // Build tree and copy it to CUDA
        constructBarnesHutTree(tree, frames[i], bodyCount);
        copyOctreeToCUDA(tree, gpuTree, capcity);

        // Copy input frame of bodies to CUDA
        hipMemcpy(gpuFrame, frames[i], bodyCount * sizeof(Body), hipMemcpyHostToDevice);

        // Simulate frame on GPU
        simulateFrame<<<(bodyCount + 255) / 256, 256>>>(gpuTree, gpuFrame, bodyCount, deltaT, theta, G);

        // Copy results back
        hipMemcpy(frames[i + 1], gpuFrame, bodyCount * sizeof(Body), hipMemcpyDeviceToHost);
    }

    // End timing
    auto endtime = std::chrono::high_resolution_clock::now();
    double runtime = std::chrono::duration_cast<std::chrono::milliseconds>(endtime - starttime).count() / 1000.0;
    printf("Simulated %d frames in %.4f seconds\n", timeSteps, runtime);

    //Put data into output format
    for(size_t i = 0; i < bodyCount; i++) {
        masses[i] = frames[0][i].mass;
    }

    for(size_t i = 0; i < timeSteps; i++) {
        for(size_t j = 0; j < bodyCount; j++) {
            pos[i][j] = frames[i][j].pos;
        }
    }

    //Write to output
    out = fopen(outputFileName, "wb");
    writeOutput(out, bodyCount, timeSteps, masses, pos);
    fclose(out);

    //Free memory
    for(size_t i = 0; i < timeSteps; i++) {
        free(frames[i]);
        free(pos[i]);
    }

    free(frames);
    free(pos);
    free(masses);
    hipFree(gpuFrame);

    freeTreeCPU(tree);
    freeTreeCUDA(gpuTree);
}