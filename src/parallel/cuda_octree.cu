#include "cuda_octree.cuh"
#include "vector_utils.cuh"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

Octree allocateOctreeCUDA(int capacity) {
    Octree tree;
    hipMalloc(&tree.children, capacity * 8 * sizeof(int));
    hipMemsetAsync(tree.children, -1, capacity * 8 * sizeof(int), 0);
    hipMalloc(&tree.centerPosition, capacity * sizeof(Vec3f));
    hipMemsetAsync(tree.centerPosition, 0, capacity * sizeof(Vec3f), 0);
    hipMalloc(&tree.massPosition, capacity * sizeof(Vec3f));
    hipMemsetAsync(tree.massPosition, 0, capacity * sizeof(Vec3f), 0);
    hipMalloc(&tree.singleBody, capacity * sizeof(int));
    hipMemsetAsync(tree.singleBody, 0, capacity * sizeof(int), 0);
    hipMalloc(&tree.mass, capacity * sizeof(float));
    hipMemsetAsync(tree.mass, 0, capacity * sizeof(float), 0);
    hipMalloc(&tree.dist, capacity * sizeof(float));
    hipMemsetAsync(tree.dist, 0, capacity * sizeof(float), 0);
    hipMalloc(&tree.nextIndex, sizeof(int));
    hipMemsetAsync(tree.nextIndex, 0, sizeof(int), 0);
    hipDeviceSynchronize();
    return tree;
}

__device__ void resetOctreeGPU(Octree tree, int capacity) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = index; i < capacity; i += blockDim.x * gridDim.x) {
        for (int j = 0; j < 8; j++)
            tree.children[i * 8 + j] = -1;
        tree.centerPosition[i] = newVec3fGPU(0, 0, 0);
        tree.massPosition[i] = newVec3fGPU(0, 0, 0);
        tree.singleBody[i] = 0;
        tree.mass[i] = 0;
        tree.dist[i] = 0;
        tree.nextIndex[i] = 0;
    }
}

void copyOctreeToCUDA(Octree cpuTree, Octree cudaTree, int capacity) {
    hipMemcpyAsync(cudaTree.children, cpuTree.children, capacity * 8 * sizeof(int), hipMemcpyHostToDevice, 0);
    hipMemcpyAsync(cudaTree.centerPosition, cpuTree.centerPosition, capacity * sizeof(Vec3f), hipMemcpyHostToDevice, 0);
    hipMemcpyAsync(cudaTree.massPosition, cpuTree.massPosition, capacity * sizeof(Vec3f), hipMemcpyHostToDevice, 0);
    hipMemcpyAsync(cudaTree.singleBody, cpuTree.singleBody, capacity * sizeof(int), hipMemcpyHostToDevice, 0);
    hipMemcpyAsync(cudaTree.mass, cpuTree.mass, capacity * sizeof(float), hipMemcpyHostToDevice, 0);
    hipMemcpyAsync(cudaTree.dist, cpuTree.dist, capacity * sizeof(float), hipMemcpyHostToDevice, 0);
    hipMemcpyAsync(cudaTree.nextIndex, cpuTree.nextIndex, sizeof(int), hipMemcpyHostToDevice, 0);
    hipDeviceSynchronize();
}

void freeTreeCUDA(Octree root) {
    hipFree(root.children);
    hipFree(root.centerPosition);
    hipFree(root.massPosition);
    hipFree(root.singleBody);
    hipFree(root.mass);
    hipFree(root.dist);
    hipFree(root.nextIndex);
}

__device__ void insertElementGPU(Octree root, int node, Vec3f newVector, float mass) {
    //If tree does not exist yet.
    if(*root.nextIndex == 0) {
        addOctreeChildGPU(root, newVector, mass, 0, 0, 0, root.maxDiameter / 2);
    }

    //Get the octant it needs to be in.
    int octant = getOctantVectorGPU(newVector, root.centerPosition[node]);
    if(root.children[node * 8 + octant] == -1) {
        //If that octant is open, just insert and return root.
        //Increase mass of overall tree, calculate new sub octree and insert it.
        float rad = root.dist[node] / 2;
        float x = root.centerPosition[node].x + rad * (newVector.x > root.centerPosition[node].x ? 1 : -1);
        float y = root.centerPosition[node].y + rad * (newVector.y > root.centerPosition[node].y ? 1 : -1);
        float z = root.centerPosition[node].z + rad * (newVector.z > root.centerPosition[node].z ? 1 : -1);
        root.children[node * 8 + octant] = addOctreeChildGPU(root, newVector, mass, x, y, z, rad);
        root.singleBody[node] = 0;
        root.massPosition[node] = newVector;
        root.mass[node] = mass;
    } else if(root.singleBody[root.children[node * 8 + octant]] == 1) {
        //When there is already a single body there, subdivide the tree.
        Body retMass = combineMassGPU(root.massPosition[node], root.mass[node], newVector, mass);
        root.massPosition[node] = retMass.pos;
        root.mass[node] = retMass.mass;
        subdivideOctreeGPU(root, root.children[node * 8 + octant], newVector, mass);
    } else {
        //If octant is not a single body, and is not null, recursive call.
        Body retMass = combineMassGPU(root.massPosition[node], root.mass[node], newVector, mass);
        root.massPosition[node] = retMass.pos;
        root.mass[node] = retMass.mass;
        insertElementGPU(root, root.children[node * 8 + octant], newVector, mass);
    }
}

__device__ int addOctreeChildGPU(Octree tree, Vec3f vector, float mass, float x, float y, float z, float dist) {
    //Get the next node
    int childIndex = *tree.nextIndex;
    *tree.nextIndex = childIndex + 1;
    //We make the center of mass hold everything of the previous body.
    //Once it is no longer a single body, that data no longer matters.
    tree.mass[childIndex] = mass;
    tree.massPosition[childIndex] = vector;
    tree.centerPosition[childIndex].x = x;
    tree.centerPosition[childIndex].y = y;
    tree.centerPosition[childIndex].z = z;
    tree.dist[childIndex] = dist;
    tree.singleBody[childIndex] = 1;
    //return the index of the new node.
    return childIndex;
}

__device__ int subdivideOctreeGPU(Octree root, int node, Vec3f newBody, float mass) {
    int flag = 1;
    int rootOctant = 0;
    int newOctant = 0;
    int useThisToReturn = node;
    do {
        flag = 0;
        //Mark it as no longer a single body.
        root.singleBody[node] = 0;
        //Get the octant of the root and new body.
        rootOctant = getOctantVectorGPU(root.massPosition[node], root.centerPosition[node]);
        newOctant = getOctantVectorGPU(newBody, root.centerPosition[node]);
        if(rootOctant != newOctant) {
            root.children[node * 8 + rootOctant] = addOctreeChildGPU(root, root.massPosition[node], root.mass[node],
                                                                  root.centerPosition[node].x, root.centerPosition[node].y, root.centerPosition[node].z, root.dist[node]);
            root.children[node * 8 + newOctant] = addOctreeChildGPU(root, newBody, mass, root.centerPosition[node].x,
                                                                 root.centerPosition[node].y, root.centerPosition[node].z, root.dist[node]);
            flag = 0;
        } else {
            float distHalf = root.dist[node] / 2;
            float x = root.centerPosition[node].x + (distHalf * (newBody.x > root.centerPosition[node].x ? 1 : -1));
            float y = root.centerPosition[node].y + (distHalf * (newBody.y > root.centerPosition[node].y ? 1 : -1));
            float z = root.centerPosition[node].z + (distHalf * (newBody.z > root.centerPosition[node].z ? 1 : -1));
            int newChild = addOctreeChildGPU(root, root.massPosition[node], root.mass[node], x, y, z, distHalf);
            root.children[node * 8 + rootOctant] = newChild;
            node = newChild;
        }

        Body retBody = combineMassGPU(root.massPosition[node], root.mass[node], newBody, mass);
        root.massPosition[node] = retBody.pos;
        root.mass[node] = retBody.mass;
    } while(flag);

    return useThisToReturn;
}

__device__ int getOctantVectorGPU(Vec3f position, Vec3f centerPosition) {
    return getOctantPositionGPU(position, centerPosition.x, centerPosition.y, centerPosition.z);
}

__device__ int getOctantPositionGPU(Vec3f position, float x, float y, float z) {
    int ret = 0;
    if(position.x >= 0) {
        ret = position.y >= x ? 1 : 4;
    } else {
        ret = position.y >= y ? 2 : 3;
    }
    ret += position.z >= z ? 0 : 4;
    return ret - 1;
}

__device__ void setDiameterGPU(Octree tree, float maxSize) {
    tree.maxDiameter = maxSize;
}
