#include "hip/hip_runtime.h"
#include "vector_utils.cuh"

__device__ Vec3f newVec3f(float x, float y, float z)
{
    Vec3f ret;
    ret.x = x;
    ret.y = y;
    ret.z = z;
    return ret;
}

__device__ float vectorDot(Vec3f u, Vec3f v)
{
    return u.x*v.x + u.y*v.y + u.z*v.z;
}

__device__ Vec3f vectorAdd(Vec3f u, Vec3f v)
{
    return newVec3f(u.x + v.x, u.y + v.y, u.z + v.z);
}

__device__ Vec3f vectorScalarMult(float f, Vec3f u)
{
    return newVec3f(t*u.x, t*u.y, t*u.z);
}

__device__ Vec3f vectorNormalize(Vec3f u)
{
    return vectorScalarMult(rnorm3df(u.x, u.y, u.z), u);
}

__device__ Vec3f ptToVector(Vec3f p1, Vec3f p2)
{
    return newVec3f(p2.x - p1.x, p2.y - p1.y, p2.z - p1.z);
}

__device__ Vec3f finalVel(Vec3f accel, Vec3f v0, double t)
{
    return newVec3f(accel.x * t + v0.x, accel.y * t + v0.y, accel.y * t + v0.y);
}

__device__ Vec3f finalPos(Vec3f accel, Vec3f v0, Vec3f p0, double t)
{
    return newVec3f(fdividef(accel.x*t*t,2) + v0.x*t + p0.x,
                    fdividef(accel.y*t*t,2) + v0.y*t + p0.y,
                    fdividef(accel.z*t*t,2) + v0.z*t + p0.z);
}

float distanceBetweenPoints(Vec3f u, Vec3f v) {
    return sqrt(u.x * v.x + u.y * v.y + u.z * v.z);

}

Body combineMass(Vec3f rootVector, float rootMass, Vec3f newVector, float newMass) {
    //Total mass is sum
    //CM = (x1m1 + x2m2)/(m1 + m2)
    Body combinedMass;
    combinedMass.mass = rootMass + newMass;
    combinedMass.pos.x = ((rootMass * rootVector.x) + (newMass * newVector.x)) / (rootMass + newMass);
    combinedMass.pos.y = ((rootMass * rootVector.y) + (newMass * newVector.y)) / (rootMass + newMass);
    combinedMass.pos.z = ((rootMass * rootVector.z) + (newMass * newVector.z)) / (rootMass + newMass);
    return combinedMass;
}

Body newEmptyBody() {
    Body b;
    b.mass = 0;
    b.pos = newVec3f(0, 0, 0);
    b.vel = newVec3f(0, 0, 0);
    return b;
}