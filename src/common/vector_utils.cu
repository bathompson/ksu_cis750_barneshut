#include "hip/hip_runtime.h"
#include "vector_utils.cuh"

__device__ Vec3f newVec3fGPU(float x, float y, float z)
{
    Vec3f ret;
    ret.x = x;
    ret.y = y;
    ret.z = z;
    return ret;
}

__device__ float vectorDotGPU(Vec3f u, Vec3f v)
{
    return u.x*v.x + u.y*v.y + u.z*v.z;
}

__device__ Vec3f vectorAddGPU(Vec3f u, Vec3f v)
{
    return newVec3fGPU(u.x + v.x, u.y + v.y, u.z + v.z);
}

__device__ Vec3f vectorScalarMultGPU(float t, Vec3f u)
{
    return newVec3fGPU(t*u.x, t*u.y, t*u.z);
}

__device__ Vec3f vectorNormalizeGPU(Vec3f u)
{
    return vectorScalarMultGPU(rnorm3df(u.x, u.y, u.z), u);
}

__device__ int vectorEqGPU(Vec3f u, Vec3f v) {
    return u.x == v.x && u.y == v.y && u.z == v.z;
}

__device__ Vec3f ptToVectorGPU(Vec3f p1, Vec3f p2)
{
    return newVec3fGPU(p2.x - p1.x, p2.y - p1.y, p2.z - p1.z);
}

__device__ Vec3f finalVelGPU(Vec3f accel, Vec3f v0, double t)
{
    return newVec3fGPU(accel.x * t + v0.x, accel.y * t + v0.y, accel.y * t + v0.y);
}

__device__ Vec3f finalPosGPU(Vec3f accel, Vec3f v0, Vec3f p0, double t)
{
    return newVec3fGPU(fdividef(accel.x*t*t,2) + v0.x*t + p0.x,
                    fdividef(accel.y*t*t,2) + v0.y*t + p0.y,
                    fdividef(accel.z*t*t,2) + v0.z*t + p0.z);
}

__device__ float distanceBetweenPointsGPU(Vec3f u, Vec3f v) {
    return sqrt(u.x * v.x + u.y * v.y + u.z * v.z);

}

__device__ Body combineMassGPU(Vec3f rootVector, float rootMass, Vec3f newVector, float newMass) {
    //Total mass is sum
    //CM = (x1m1 + x2m2)/(m1 + m2)
    Body combinedMass;
    combinedMass.mass = rootMass + newMass;
    combinedMass.pos.x = ((rootMass * rootVector.x) + (newMass * newVector.x)) / (rootMass + newMass);
    combinedMass.pos.y = ((rootMass * rootVector.y) + (newMass * newVector.y)) / (rootMass + newMass);
    combinedMass.pos.z = ((rootMass * rootVector.z) + (newMass * newVector.z)) / (rootMass + newMass);
    return combinedMass;
}

__device__ Body newEmptyBodyGPU() {
    Body b;
    b.mass = 0;
    b.pos = newVec3fGPU(0, 0, 0);
    b.vel = newVec3fGPU(0, 0, 0);
    return b;
}